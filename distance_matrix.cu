#include "hip/hip_runtime.h"
// Compute the distance matrix (only in the upper diagonal)
__global__ void ComputeDistanceMatrix(float Data[N][M],float DistMatrix[M][M], int blockIncr)
{
  // Ci sono problemi di bordo
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int i,j;
  if(by > bx) {
    i = (bx + blockIncr) * blockDim.x + threadIdx.x;
    j = (by + blockIncr) * blockDim.y + threadIdx.y;
  } else if(bx > by) {
    i = bx * blockDim.x + threadIdx.x;
    j = by * blockDim.y + threadIdx.y;
  } else {
    if(threadIdx.y > threadIdx.x) {
      /*  If I am in the lower triangle, simply transpose the index (after moving
          to the lower part of the matrix)
      */
      j = (bx + blockIncr) * blockDim.x + threadIdx.x;
      i = (by + blockIncr) * blockDim.y + threadIdx.y;
    } else {
      i = bx * blockDim.x + threadIdx.x;
      j = by * blockDim.y + threadIdx.y;
    }
  }
  if (i==j) {
    DistMatrix[i][j] = 0;
    return;
  } else if (i > M || j > M){
    return;
  }

  // Compute the Euclidian distance between points i and j
  int span;
  float sqDist = 0;
  for(span = 0; span < N; span++) {
    float diff = Data[span][i] - Data[span][j];
    sqDist += diff * diff;
  }
  float dist = sqrtf(sqDist);
  DistMatrix[i][j] = dist;
}

int main()
{
  // Memory Allocation etc. goes here.

  //check rounding
  dim3 threadsPerBlock(16,16);
  dim3 numBlocks(M / threadsPerBlock.x, M / (2 * threadsPerBlock.y));
  ComputeDistanceMatrix<<<numBlocks, threadsPerBlock>>>(Data, DistMatrix,numBlocks.y);
}
